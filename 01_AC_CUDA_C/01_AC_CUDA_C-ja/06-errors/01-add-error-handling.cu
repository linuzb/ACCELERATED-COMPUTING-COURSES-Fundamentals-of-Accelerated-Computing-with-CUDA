
#include <hip/hip_runtime.h>
#include <stdio.h>

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

__global__
void doubleElements(int *a, int N)
{

  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = gridDim.x * blockDim.x;

  for (int i = idx; i < N + stride; i += stride)
  {
    a[i] *= 2;
  }
}

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  /*
   * エラーの内容を確認して修正するために、このソース コードに
   * エラー処理を追加します。エラーの解決方法が不明な場合は
   * Google でエラー メッセージを検索してみてください。
   */

  int N = 10000;
  int *a;

  size_t size = N * sizeof(int);
  hipMallocManaged(&a, size);

  init(a, N);

  size_t threads_per_block = 2048;
  size_t number_of_blocks = 32;

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  hipFree(a);
}
