
#include <hip/hip_runtime.h>
__global__
void d(int *a, int N)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = idx; i < N; i += stride)
  {
    a[i] = 1;
  }
}

void h(int *a, int N)
{
  for (int i = 0; i < N; ++i)
  {
    a[i] = 1;
  }
}

int main()
{
  int N = 2<<24;
  size_t size = N * sizeof(int);
  int *a;
  hipMallocManaged(&a, size);
  d<<<256, 256>>>(a, N);
  hipFree(a);
}
